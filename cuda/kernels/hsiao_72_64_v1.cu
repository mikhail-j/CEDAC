
#include <hip/hip_runtime.h>
/* Encode and decode using the Hsiao(72, 64) version 1 code in CUDA
 *
 * Copyright (C) 2020 Qijia (Michael) Jin
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

extern "C" {

__device__ void fix_single_bit_error_hsiao_72_64_v1(unsigned long long* data, unsigned char* parity) {
	unsigned long long new_data;
#pragma unroll
	for (int i = 0; i < 64; i++) {
		new_data = (*data) ^ (1ULL << i);
		if (!(((__popcll(0x0738c808099264ff & new_data) & 1) ^ (__popc(0x01 & (*parity)) & 1))
			| ((__popcll(0x38c808099264ff07 & new_data) & 1) ^ (__popc(0x02 & (*parity)) & 1))
			| ((__popcll(0xc808099264ff0738 & new_data) & 1) ^ (__popc(0x04 & (*parity)) & 1))
			| ((__popcll(0x08099264ff0738c8 & new_data) & 1) ^ (__popc(0x08 & (*parity)) & 1))
			| ((__popcll(0x099264ff0738c808 & new_data) & 1) ^ (__popc(0x10 & (*parity)) & 1))
			| ((__popcll(0x9264ff0738c80809 & new_data) & 1) ^ (__popc(0x20 & (*parity)) & 1))
			| ((__popcll(0x64ff0738c8080992 & new_data) & 1) ^ (__popc(0x40 & (*parity)) & 1))
			| ((__popcll(0xff0738c808099264 & new_data) & 1) ^ (__popc(0x80 & (*parity)) & 1)))) {
			*data = new_data;
			return;
		}
	}
	unsigned char new_parity;
#pragma unroll
	for (int i = 0; i < 8; i++) {
		new_parity = (*parity) ^ (1 << i);
		if (!(((__popcll(0x0738c808099264ff & (*data)) & 1) ^ (__popc(0x01 & new_parity) & 1))
			| ((__popcll(0x38c808099264ff07 & (*data)) & 1) ^ (__popc(0x02 & new_parity) & 1))
			| ((__popcll(0xc808099264ff0738 & (*data)) & 1) ^ (__popc(0x04 & new_parity) & 1))
			| ((__popcll(0x08099264ff0738c8 & (*data)) & 1) ^ (__popc(0x08 & new_parity) & 1))
			| ((__popcll(0x099264ff0738c808 & (*data)) & 1) ^ (__popc(0x10 & new_parity) & 1))
			| ((__popcll(0x9264ff0738c80809 & (*data)) & 1) ^ (__popc(0x20 & new_parity) & 1))
			| ((__popcll(0x64ff0738c8080992 & (*data)) & 1) ^ (__popc(0x40 & new_parity) & 1))
			| ((__popcll(0xff0738c808099264 & (*data)) & 1) ^ (__popc(0x80 & new_parity) & 1)))) {
			*parity = new_parity;
			return;
		}
	}
	return;
}

__device__ void syndrome_decoding_hsiao_72_64_v1(unsigned long long* data, unsigned char* parity, unsigned long long* errors) {
	if (((__popcll(0x0738c808099264ff & (*data)) & 1) ^ (__popc(0x01 & (*parity)) & 1))
			| ((__popcll(0x38c808099264ff07 & (*data)) & 1) ^ (__popc(0x02 & (*parity)) & 1))
			| ((__popcll(0xc808099264ff0738 & (*data)) & 1) ^ (__popc(0x04 & (*parity)) & 1))
			| ((__popcll(0x08099264ff0738c8 & (*data)) & 1) ^ (__popc(0x08 & (*parity)) & 1))
			| ((__popcll(0x099264ff0738c808 & (*data)) & 1) ^ (__popc(0x10 & (*parity)) & 1))
			| ((__popcll(0x9264ff0738c80809 & (*data)) & 1) ^ (__popc(0x20 & (*parity)) & 1))
			| ((__popcll(0x64ff0738c8080992 & (*data)) & 1) ^ (__popc(0x40 & (*parity)) & 1))
			| ((__popcll(0xff0738c808099264 & (*data)) & 1) ^ (__popc(0x80 & (*parity)) & 1))) {
		if (((__popcll(0x0738c808099264ff & (*data)) & 1) ^ (__popc(0x01 & (*parity)) & 1))
			^ ((__popcll(0x38c808099264ff07 & (*data)) & 1) ^ (__popc(0x02 & (*parity)) & 1))
			^ ((__popcll(0xc808099264ff0738 & (*data)) & 1) ^ (__popc(0x04 & (*parity)) & 1))
			^ ((__popcll(0x08099264ff0738c8 & (*data)) & 1) ^ (__popc(0x08 & (*parity)) & 1))
			^ ((__popcll(0x099264ff0738c808 & (*data)) & 1) ^ (__popc(0x10 & (*parity)) & 1))
			^ ((__popcll(0x9264ff0738c80809 & (*data)) & 1) ^ (__popc(0x20 & (*parity)) & 1))
			^ ((__popcll(0x64ff0738c8080992 & (*data)) & 1) ^ (__popc(0x40 & (*parity)) & 1))
			^ ((__popcll(0xff0738c808099264 & (*data)) & 1) ^ (__popc(0x80 & (*parity)) & 1))) {
			fix_single_bit_error_hsiao_72_64_v1(data, parity);
			atomicAdd(errors, 1ULL);
			return;
		}
		else {
			atomicAdd(errors + 1, 1ULL);
			return;
		}
	}
	return;
}

__global__ void generate_parity_hsiao_72_64_v1(unsigned long long* data, unsigned char* parity, size_t data_size) {
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < data_size) {
		parity[i] = ((__popcll(0x0738c808099264ff & data[i]) & 1)
			+ ((__popcll(0x38c808099264ff07 & data[i]) & 1) << 1)
			+ ((__popcll(0xc808099264ff0738 & data[i]) & 1) << 2)
			+ ((__popcll(0x08099264ff0738c8 & data[i]) & 1) << 3)
			+ ((__popcll(0x099264ff0738c808 & data[i]) & 1) << 4)
			+ ((__popcll(0x9264ff0738c80809 & data[i]) & 1) << 5)
			+ ((__popcll(0x64ff0738c8080992 & data[i]) & 1) << 6)
			+ ((__popcll(0xff0738c808099264 & data[i]) & 1) << 7));
		i = i + (blockDim.x * gridDim.x);
	}

	return;
}

__global__ void edac_hsiao_72_64_v1(unsigned long long* data, unsigned char* parity, size_t data_size, unsigned long long* error_count) {
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < data_size) {
		syndrome_decoding_hsiao_72_64_v1(data + i, parity + i, error_count);
		i = i + (blockDim.x * gridDim.x);
	}

	return;
}

}
