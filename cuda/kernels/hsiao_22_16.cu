
#include <hip/hip_runtime.h>
/* Encode and decode using the Hsiao(22, 16) code in CUDA
 *
 * Copyright (C) 2020 Qijia (Michael) Jin
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */

extern "C" {

__device__ void fix_single_bit_error_hsiao_22_16(unsigned short* data, unsigned char* parity) {
	unsigned short new_data;
#pragma unroll
	for (int i = 0; i < 16; i++) {
		new_data = (*data) ^ (((unsigned short)1) << i);
		if (!(((__popc(0x443f & new_data) & 1) ^ (__popc(0x01 & (*parity)) & 1))
			| ((__popc(0x13c7 & new_data) & 1) ^ (__popc(0x02 & (*parity)) & 1))
			| ((__popc(0xe1d1 & new_data) & 1) ^ (__popc(0x04 & (*parity)) & 1))
			| ((__popc(0xee60 & new_data) & 1) ^ (__popc(0x08 & (*parity)) & 1))
			| ((__popc(0x3e8a & new_data) & 1) ^ (__popc(0x10 & (*parity)) & 1))
			| ((__popc(0x993c & new_data) & 1) ^ (__popc(0x20 & (*parity)) & 1)))) {
			*data = new_data;
			return;
		}
	}
	unsigned char new_parity;
#pragma unroll
	for (int i = 0; i < 6; i++) {
		new_parity = (*parity) ^ (((unsigned char)1) << i);
		if (!(((__popc(0x443f & (*data)) & 1) ^ (__popc(0x01 & new_parity) & 1))
			| ((__popc(0x13c7 & (*data)) & 1) ^ (__popc(0x02 & new_parity) & 1))
			| ((__popc(0xe1d1 & (*data)) & 1) ^ (__popc(0x04 & new_parity) & 1))
			| ((__popc(0xee60 & (*data)) & 1) ^ (__popc(0x08 & new_parity) & 1))
			| ((__popc(0x3e8a & (*data)) & 1) ^ (__popc(0x10 & new_parity) & 1))
			| ((__popc(0x993c & (*data)) & 1) ^ (__popc(0x20 & new_parity) & 1)))) {
			*parity = new_parity;
			return;
		}
	}
	return;
}

__device__ void syndrome_decoding_hsiao_22_16(unsigned short* data, unsigned char* parity, unsigned long long* errors) {
	if (((__popc(0x443f & (*data)) & 1) ^ (__popc(0x01 & (*parity)) & 1))
			| ((__popc(0x13c7 & (*data)) & 1) ^ (__popc(0x02 & (*parity)) & 1))
			| ((__popc(0xe1d1 & (*data)) & 1) ^ (__popc(0x04 & (*parity)) & 1))
			| ((__popc(0xee60 & (*data)) & 1) ^ (__popc(0x08 & (*parity)) & 1))
			| ((__popc(0x3e8a & (*data)) & 1) ^ (__popc(0x10 & (*parity)) & 1))
			| ((__popc(0x993c & (*data)) & 1) ^ (__popc(0x20 & (*parity)) & 1))) {
		if (((__popc(0x443f & (*data)) & 1) ^ (__popc(0x01 & (*parity)) & 1))
			^ ((__popc(0x13c7 & (*data)) & 1) ^ (__popc(0x02 & (*parity)) & 1))
			^ ((__popc(0xe1d1 & (*data)) & 1) ^ (__popc(0x04 & (*parity)) & 1))
			^ ((__popc(0xee60 & (*data)) & 1) ^ (__popc(0x08 & (*parity)) & 1))
			^ ((__popc(0x3e8a & (*data)) & 1) ^ (__popc(0x10 & (*parity)) & 1))
			^ ((__popc(0x993c & (*data)) & 1) ^ (__popc(0x20 & (*parity)) & 1))) {
			fix_single_bit_error_hsiao_22_16(data, parity);
			atomicAdd(errors, 1ULL);
			return;
		}
		else {
			atomicAdd(errors + 1, 1ULL);
			return;
		}
	}
	return;
}

__global__ void generate_parity_hsiao_22_16(unsigned short* data, unsigned char* parity, size_t data_size) {
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < data_size) {
		parity[i] = ((__popc(0x443f & data[i]) & 1)
			+ ((__popc(0x13c7 & data[i]) & 1) << 1)
			+ ((__popc(0xe1d1 & data[i]) & 1) << 2)
			+ ((__popc(0xee60 & data[i]) & 1) << 3)
			+ ((__popc(0x3e8a & data[i]) & 1) << 4)
			+ ((__popc(0x993c & data[i]) & 1) << 5));
		i = i + (blockDim.x * gridDim.x);
	}

	return;
}

__global__ void edac_hsiao_22_16(unsigned short* data, unsigned char* parity, size_t data_size, unsigned long long* error_count) {
	size_t i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < data_size) {
		syndrome_decoding_hsiao_22_16(data + i, parity + i, error_count);
		i = i + (blockDim.x * gridDim.x);
	}

	return;
}

}
